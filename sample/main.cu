
#include <hip/hip_runtime.h>
#include <iostream>

// CUDAカーネル関数
__global__ void helloWorldKernel() {
    printf("Hello, World from GPU!\n");
}

int main() {
    // GPU上でカーネルを実行
    helloWorldKernel<<<10, 10>>>();

    // CUDAのデバイス同期を待機
    hipDeviceSynchronize();

    // ホスト側のメッセージ
    std::cout << "Hello, World from CPU!" << std::endl;

    return 0;
}